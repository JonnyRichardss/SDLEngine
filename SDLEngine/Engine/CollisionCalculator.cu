#include "hip/hip_runtime.h"
#include "CollisionCalculator.cuh"
#include "hip/hip_runtime.h"
#include ""
//#include "Global_Flags.h" TODO!
#include "GameMath.h"
#include <math.h>
#include "GameLogging.h"
static int MAX_COLLISIONS = 100;
struct GameObjectCUDA {
	int id;
	/*
	* points MUST be fed as
	* (technically order doesnt matter but 0 and 3 must be opposite)
	0-------1
	|       |
	|       |
	2-------3
	*/
	float2 points[4];
	float2 centre;
	int colliders[MAX_COLLISIONS];
	float2 collisionVecs[MAX_COLLISIONS];
	unsigned int currentIdx = 0;
};
static GameObjectCUDA* objsGPU;
static bool allocated = false;
__device__ inline float dotProduct(float2 a, float2 b) {
	return (a.x * b.x) + (a.y * b.y);
}
__device__ inline float2 CalcNormalAxis(float2 p1, float2 p2) {
	//return -dy, dx
	float dy = p1.y - p2.y;
	float dx = p1.x - p2.x;
	float length = sqrtf(dx * dx + dy * dy);
	float2 axis = make_float2(dy / (length * -1), dx / length);
	return axis;
}
__device__ inline float2 MinMaxProjectionOnAxis(float2 points[4],float2 axis) {
	//https://textbooks.cs.ksu.edu/cis580/04-collisions/04-separating-axis-theorem/index.html
	//accessed 19/04/24
	//project each point
	//return min,max
	float projection = dotProduct(points[0],axis);
	float min = projection;
	float max = projection;
	for (int i = 1; i < 4; i++) {
		projection = dotProduct(points[i], axis);
		max = max > projection ? max : projection;
		min = min < projection ? min : projection;
	}
	return make_float2(min, max);
}
__device__ inline float Magnitude(float2 vec) {
	return sqrt((vec.x * vec.x)+(vec.y * vec.y));
}
__device__ inline bool SATCheck(GameObjectCUDA* object, GameObjectCUDA* other,float2& vec,float& vecMag) {
	//TRUE IS A COLLISION
	//Last step -- implement actual alg
	//we are assuming boxes so we only need up normal and right normal for both
	//i added all of them in when debugging but i dont think they are needed
	float2 normals[8];
	normals[0] = CalcNormalAxis(object->points[0], object->points[1]);
	normals[1] = CalcNormalAxis(object->points[1], object->points[3]);
	normals[2] = CalcNormalAxis(object->points[0], object->points[2]);
	normals[3] = CalcNormalAxis(object->points[2], object->points[3]);
	normals[4] = CalcNormalAxis(other->points[0], other->points[1]);
	normals[5] = CalcNormalAxis(other->points[1], other->points[3]);
	normals[6] = CalcNormalAxis(other->points[0], other->points[2]);
	normals[7] = CalcNormalAxis(other->points[2], other->points[3]);
	float minOverlap = FLT_MAX;
	for (int i = 0; i < 4; i++) {
		float2 objectProjection = MinMaxProjectionOnAxis(object->points, normals[i]);
		float2 otherProjection = MinMaxProjectionOnAxis(other->points, normals[i]);
		float2 overlap = make_float2(otherProjection.y - objectProjection.x , objectProjection.y-otherProjection.x);
		if (overlap.x < 0 ||overlap.y < 0) {
			//there is no overlap
			return false;
		}

		float mag = (overlap.x < overlap.y) ? overlap.x : overlap.y;
		if (mag < minOverlap) {
			vec = normals[i];
			vecMag = mag;
		}
	}

	return true;
}
__device__ float dot(float2 a, float2 b) {
	return (a.x * b.x) + (a.y * b.y);
}
__global__ void GPUCollisionCalc(GameObjectCUDA* objs, int size) {
	int i = blockIdx.x;
	if (i > size) return;//think this is never true but oh well
	for (int j = threadIdx.x; j < size; j += blockDim.x) {
		if (i == j) continue;
		GameObjectCUDA* object = &objs[i];
		GameObjectCUDA* other = &objs[j];
		//TRUE IS A COLLISION

		//if collision found add to j.id to colliders
		float2 collisionVec;
		float collisionVecMag;
		if (SATCheck(object, other,collisionVec,collisionVecMag)) {
			unsigned int index = atomicInc(&objs[i].currentIdx, (MAX_COLLISIONS));//atomic inc prevents race conditions - each thread *should* always have its own unique index to access
			objs[i].colliders[index] = objs[j].id;
			float2 offset = make_float2(other->centre.x - object->centre.x, other->centre.y - object->centre.y);
			collisionVec = make_float2(collisionVec.x * collisionVecMag,collisionVec.y * collisionVecMag);
			if (dot(offset, collisionVec) > 0) {
				objs[i].collisionVecs[index] = make_float2(0 - collisionVec.x, 0 - collisionVec.y);
			}
			else {
				objs[i].collisionVecs[index] = collisionVec;
			}
		}
	}
}
void MakePoints(JRrect rect,GameObjectCUDA& output) {
	for (int i = 0; i < 4; i++) {
		output.points[i] = make_float2(rect.points[i].x, rect.points[i].y);
	}
}
void MakeStructs(GameObjectCUDA* output, std::vector<GameObject*>& input) {
	for (int i = 0; i < input.size(); i++) {
		//clear colliders from last frame
		input[i]->colliders.clear();
		output[i].id = i;
		Vector2 pos = input[i]->GetPos();
		output[i].centre = make_float2(pos.x,pos.y);
		MakePoints(input[i]->GetCorners(), output[i]);
		input[i]->colliders.clear();
		input[i]->collisionVectors.clear();
	}
}
void UnMakeStructs(std::vector<GameObject*>& output, GameObjectCUDA* input) {
	for (int i = 0; i < output.size(); i++) {
		int idx = static_cast<int>(input[i].currentIdx);
		for (int j = 0; j < std::min(idx, (int)output.size()); j++) {
			if (i == input[i].colliders[j])
				continue;
			if (output[i] == nullptr) 
				continue;
			int outID = input[i].colliders[j];
			if (outID < 0 || outID >= output.size()) {
				//GameLogging::GetInstance()->Log("ERROR: Invalid ID for detected collider (objectID colliderID) " + std::to_string(i) + " " + std::to_string(outID));
				continue;
			}

			output[i]->colliders.push_back(output[outID]);
			float2 CollisionVec = input[i].collisionVecs[j];
			Vector2 CollisionVec2 = Vector2(CollisionVec.x, CollisionVec.y);
			output[i]->collisionVectors.push_back(CollisionVec2);
		}
	}
}
void FreeCUDA() {
	if (allocated)
		hipFree(objsGPU);
	allocated = false;
}
void AllocCUDA(int size) {
	if (allocated)
		FreeCUDA();
	hipMalloc((void**)&objsGPU, sizeof(GameObjectCUDA) * size);
	allocated = true;
}
namespace JRCollision {
	void Init() {
		//current state has this immediately reallocated
		//AllocCUDA(AllocSize); TODO!?
	}
	void Free()
	{
		FreeCUDA();
	}
	void CalculateCollsion(std::vector<GameObject*>& UpdateQueue)
	{
		//make sure we have enough room on the GPU -- doing it this way prevents us from allocating every frame for different update queue sizes
		/*
		bool need_new_alloc = false;
		while (AllocSize < UpdateQueue.size()) {
			AllocSize += COLLISION_EXPAND_SIZE;
			need_new_alloc = true;
		}
		if (need_new_alloc || !allocated) {
			AllocCUDA(AllocSize);
		}
		*/
		AllocCUDA(UpdateQueue.size()); // NOTE TO SELF  -- THIS WANTS FIXING -- MAYBE WONT IMPACT PERFORMANCE BUT IT ANNOYS ME
		GameObjectCUDA* objs = new GameObjectCUDA[UpdateQueue.size()];
		//copy to
		MakeStructs(objs, UpdateQueue);
		hipMemcpy(objsGPU, objs, sizeof(GameObjectCUDA) * UpdateQueue.size(), hipMemcpyHostToDevice);

		//exec
		GPUCollisionCalc << < UpdateQueue.size(), 1024 >> > (objsGPU, UpdateQueue.size());
		//copy from
		hipMemcpy(objs, objsGPU, sizeof(GameObjectCUDA) * UpdateQueue.size(), hipMemcpyDeviceToHost);
		UnMakeStructs(UpdateQueue, objs);
		//free
		delete[] objs;
		FreeCUDA();
	}
}