#include "hip/hip_runtime.h"
#include "Boid.cuh"
#include "SDL.h"
#include "SDL_image.h"
#include "GameRNG.h"
#include "BoidScene.h"
#include <math.h>


//boids consts
static constexpr float BOID_SIZE = 10.0f;
static constexpr float BOID_SPEED = 4.0f;

static constexpr int NEIGHBOUR_STORAGE_DIV = 1; //divisor to reduce size of neighbour array - deemed unnecessary as it *appears* that memcpy has O(1) like performance (maybe not but at a glance there is no benefit to copying less data at this scale)
static constexpr float BOID_VISION_DISTANCE = 45.0f;
static constexpr float BOID_AVOID_DISTANCE = BOID_VISION_DISTANCE;// will try change later
static constexpr float BOID_VISION_ANGLE = 3.141592653589;
static constexpr float BOID_STEER_MULTIPLIER = 0.07f;
static constexpr float BOID_SEPARATION_STRENGTH = 1.0f;
static constexpr float BOID_ALIGNMENT_STRENGTH = 0.3f;
static constexpr float BOID_COHESION_STRENGTH = 1.0f;
static constexpr float RAND_WALK_SIZE_RAD = 0.05f;
void Boid::Init()
{
	isBoid = true;
	BoundingBox = Vector2(BOID_SIZE, BOID_SIZE * 2);
	shown = true;
	facing = RNG::randf(0, 2 * M_PI);
	velocity = Vector2::zero();
	position = Vector2(RNG::randi(0, GAME_MAX_X * 2) - GAME_MAX_X, RNG::randi(0, GAME_MAX_Y * 2) - GAME_MAX_Y); //maybe ill eventually fix my random functions to behave with -ve values
	DoRotation();
	steerTarget = Vector2::zero();
}
void Boid::SetOwner(BoidScene* owner){
	manager = owner;
	GameScene* newPtr = dynamic_cast<GameScene*>(owner);
	if (newPtr != nullptr)
		GameObject::SetOwner(newPtr);
}
void Boid::SetName(int num)
{
	name = "Boid " + std::to_string(num);
}
void Boid::InitVisuals()
{
	visuals->LoadTexture("boid", ".png");
	SDL_Texture* Tex = visuals->GetTexture();
	SDL_SetTextureColorMod(Tex, RNG::randi(0, 0), RNG::randi(0, 128), RNG::randi(128, 255));
	SDL_Rect DefaultRect = BBtoDestRect();
	visuals->UpdateDestPos(&DefaultRect);
	//visuals->UpdateLayer(69420);
	//visuals->UpdateFlip(static_cast<SDL_RendererFlip>(SDL_FLIP_VERTICAL | SDL_FLIP_HORIZONTAL));

}

bool Boid::Update()
{
	numNeighbours = Neighbours.size();
	steerTarget = Vector2::zero();
	CalcTarget();
	SteerTowards(steerTarget);
	ScreenWrap();
	DoRotation();
	Neighbours.clear();
	return true;
}

void Boid::DoRotation()
{
	 velocity = Vector2(sin(facing) * BOID_SPEED, cos(facing) * BOID_SPEED);
}

void Boid::ScreenWrap()
{
	if (position.x > GAME_MAX_X) {
		position.x = -GAME_MAX_X;
	}
	else if (position.x < -GAME_MAX_X) {
		position.x = GAME_MAX_X;
	}
	if (position.y > GAME_MAX_Y) {
		position.y = -GAME_MAX_Y;
	}
	else if (position.y < -GAME_MAX_Y) {
		position.y = GAME_MAX_Y;
	}
}

void Boid::CalcTarget()
{
	Vector2 aligVec = Vector2::zero();
	Vector2 sepVec = Vector2::zero();
	Vector2 cohesVec = Vector2::zero();
	for (Boid* b : Neighbours) {

		aligVec -= b->GetVelo();
		cohesVec += b->GetPos();
		sepVec -= (position - b->position);
	}
	DoSeparation(sepVec);
	DoAlignment(aligVec);
	DoCohesion(cohesVec);

}

void Boid::SteerTowards(Vector2 target)
{
	facing += Vector2::AngleBetweenRAD(velocity, target) * BOID_STEER_MULTIPLIER;
	facing += (RNG::randf(0, RAND_WALK_SIZE_RAD * 2) - RAND_WALK_SIZE_RAD);
	//actually steer in velocity thanks


}
void Boid::DoSeparation(Vector2 vec)
{
	if (numNeighbours == 0)
		return;
	vec *= (1.0 / numNeighbours);
	vec *= BOID_SEPARATION_STRENGTH;
	steerTarget += vec;
}
void Boid::DoAlignment(Vector2 vec)
{
	if (numNeighbours == 0)
		return;
	vec = vec.Normalise();
	vec *= BOID_ALIGNMENT_STRENGTH;
	steerTarget += vec;
}
void Boid::DoCohesion(Vector2 vec)
{
	if (numNeighbours == 0)
		return;
	vec *= (1.0 / numNeighbours);
	vec = position - vec;
	vec *= BOID_COHESION_STRENGTH;
	steerTarget += vec;
}

namespace JRCudaCalc {
#include "hip/hip_runtime.h"
#include ""
	constexpr int NUMBLOCKS = NUM_BOIDS;
	constexpr int NUMTHREADS = 1024;
	struct CudaNeighbourStruct {
		int id;
		float2 position;
		int neighbours[NUM_BOIDS];
		//int neighbours[1024];
		unsigned int currentIdx = 0;
	};
	//(as from my previous AMP code) most of this adapted from
	//https://github.com/SebLague/Boids/blob/master/Assets/Scripts/BoidCompute.compute
	//not anymore but *some* basics of the way it's calculated on the CPU are
	__global__ void GPUNeighbourCalc(CudaNeighbourStruct* boids, int size, float sqrVisDist) {
		int itest = blockIdx.x;

		if (itest > size) return;//think this is never true but oh well
		for (int jtest = threadIdx.x; jtest < size; jtest += blockDim.x) {
			float2 offset = make_float2(boids[itest].position.x - boids[jtest].position.x, boids[itest].position.y - boids[jtest].position.y);
			float sqrDist = offset.x * offset.x + offset.y * offset.y;
			if (sqrDist < sqrVisDist) {
				unsigned int index = atomicInc(&boids[itest].currentIdx, (NUM_BOIDS));
				boids[itest].neighbours[index] = boids[jtest].id;
			}
		}
	}
	void MakeNeighbourStructs(CudaNeighbourStruct* output, std::vector<GameObject*>& input) {
		int size = input.size();
		for (int i = 0; i < size; i++) {
			Boid* boidPtr = dynamic_cast<Boid*>(input[i]);
			if (boidPtr == nullptr) continue;
			Vector2 position = boidPtr->GetPos();
			output[i].id = i;
			output[i].position = make_float2(position.x, position.y);
		}
	}
	void UnMakeNeighbourStructs(std::vector<GameObject*>& output, CudaNeighbourStruct* input) {
		int size = output.size();
		for (int i = 0; i < size; i++) {
			Boid* iBoidPtr = dynamic_cast<Boid*>(output[i]);
			if (iBoidPtr == nullptr) continue;
			int idx = static_cast<int>(input[i].currentIdx);
			for (int j = 0; j < std::min(idx, NUM_BOIDS); j++) {
				if (i == input[i].neighbours[j])continue;
				Boid* jBoidPtr = dynamic_cast<Boid*>(output[input[i].neighbours[j]]);
				if (jBoidPtr == nullptr) continue;
				iBoidPtr->Neighbours.push_back(jBoidPtr);
			}
		}
	}
	static CudaNeighbourStruct* gpuBoids;
	static size_t arraySize;
	void GetNeighboursCUDA(std::vector<GameObject*>& AllBoids)
	{
		//GameLogging::GetInstance()->DebugLog("OI");
		int size = AllBoids.size();
		CudaNeighbourStruct* boids = new CudaNeighbourStruct[size];
		MakeNeighbourStructs(boids, AllBoids);
		//allocate and copy
		//CudaBoidStruct* gpuBoids;

		hipMemcpy(gpuBoids, boids, arraySize, hipMemcpyHostToDevice);
		//do calc
		float sqrVisDist = BOID_VISION_DISTANCE * BOID_VISION_DISTANCE;
		float sqrAvoidDist = BOID_AVOID_DISTANCE * BOID_AVOID_DISTANCE;
		GPUNeighbourCalc << <NUMBLOCKS, NUMTHREADS >> > (gpuBoids, size, sqrVisDist);
		//copy back
		hipMemcpy(boids, gpuBoids, arraySize, hipMemcpyDeviceToHost);
		UnMakeNeighbourStructs(AllBoids, boids);
		//free all

		delete[] boids;
	}
	void Alloc(int size) {
		arraySize = sizeof(CudaNeighbourStruct) * size;
		hipMalloc((void**)&gpuBoids, arraySize);
	}
	void Free() {
		hipFree(gpuBoids);
	}

}


